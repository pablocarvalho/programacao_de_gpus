


#include <hip/hip_runtime.h>
#include <stdio.h>
	__global__ void add(int* a, int* b, int* c, int n)
	{

	    int id = threadIdx.x;	    

	    if(id < n )
	        c[id] = a[id] + b[id];
	}


	int main(void) {
	    int n = 1000;

	    int* a;
	    int* b;
	    int* c;

	    size_t nbytes = n * sizeof(int);

	    hipMallocManaged (&a, nbytes);
        hipMallocManaged (&b, nbytes);
        hipMallocManaged (&c, nbytes);
	   
	    for(int i = 0; i < n; i++)
	    {
	        a[i] = 1;
	        b[i] = 2;
	    }	    

        add<<<1,n>>>(a,b,c,n);
        

        hipDeviceSynchronize();

        int sum = 0;
        for(int i = 0; i < n; i++)
        {
        	sum+=c[i];
        }
        printf("%d\n",sum);

        hipFree(&a);
        hipFree(&b);
        hipFree(&c);

        return 0;



	}


